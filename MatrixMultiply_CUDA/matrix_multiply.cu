#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
// following code illustrates the gpu kernel for matrix multiplication...
__global__ void matrixMultiply(int *A, int *B, int *C, int M, int N, int P)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row index of C
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column index of C

    if (row < M && col < P) {
        int sum = 0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * P + col];
        }
        C[row * P + col] = sum;
    }
}

// function ,, to launch matrix multiplication kernel
void launchMatrixMultiply(int *A, int *B, int *C, int M, int N, int P)
{
    int *d_A, *d_B, *d_C;

    // allocate memory on the GPU
    hipMalloc(&d_A, M * N * sizeof(int));
    hipMalloc(&d_B, N * P * sizeof(int));
    hipMalloc(&d_C, M * P * sizeof(int));

    // copy data from host to device
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * P * sizeof(int), hipMemcpyHostToDevice);

    // define block and grid size
    dim3 blockSize(16, 16); // 16x16 threads per block
    dim3 gridSize((P + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    // Launch the matrix multiply kernel
    matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, P);

    // Check for errors in kernel launch
    hipDeviceSynchronize();

    // Copy the result matrix back to the host
    hipMemcpy(C, d_C, M * P * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    // Define the dimensions of the matrices
    int M = 3;  // Rows in A and C
    int N = 3;  // Columns in A and Rows in B
    int P = 3;  // Columns in B and C

    // allocate memory for matrices A, B, and C
    int A[M * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int B[N * P] = {9, 8, 7, 6, 5, 4, 3, 2, 1};
    int C[M * P]; // Result matrix

    // launch matrix multiplication on GPU
    launchMatrixMultiply(A, B, C, M, N, P);

    // / print the result
    std::cout << "Matrix C (Result):\n";
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++) {
            std::cout << C[i * P + j] << " ";
        }
        std::cout << "\n";
    }

    return 0;
}
